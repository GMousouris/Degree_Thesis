#include "hip/hip_runtime.h"
#pragma once

#include <optixu/optixu_math_namespace.h>
#include "common.h"
#include "helpers.h"
#include "per_ray_data.h"
#include "light_structs.h"

#include "sampling_functions.h"
#include "random_number_generators.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(float3, sys_CameraPosition, , );
rtDeclareVariable(float3, sys_CameraU, , );
rtDeclareVariable(float3, sys_CameraV, , );
rtDeclareVariable(float3, sys_CameraW, , );
rtDeclareVariable(float, sys_SceneEpsilon, , );

rtBuffer<ParallelogramLight> lights;
rtBuffer<float4, 2>              sys_OutputBuffer;
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

rtDeclareVariable(PerRayData_radiance, current_prd, rtPayload, );
rtDeclareVariable(unsigned int, max_depth, ,);
rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );


//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------
RT_PROGRAM void path_trace()
{
	
	size_t2 screen = sys_OutputBuffer.size();

	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

	float2 jitter_scale = inv_screen / sqrt_num_samples;
	unsigned int samples_per_pixel = sqrt_num_samples * sqrt_num_samples;
	float3 result = make_float3(0.0f);

	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, frame_number);
	do
	{
		
		//
		// Sample pixel using jittering
		//
		unsigned int x = samples_per_pixel % sqrt_num_samples;
		unsigned int y = samples_per_pixel / sqrt_num_samples;
		float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
		float2 d = pixel + jitter * jitter_scale;
		float3 ray_origin = sys_CameraPosition;
		float3 ray_direction = normalize(d.x*sys_CameraU + d.y*sys_CameraV + sys_CameraW);

		// Initialze per-ray data
		PerRayData_radiance prd;
		prd.result = make_float3(0.f);
		prd.attenuation = make_float3(1.f);
		prd.countEmitted = true;
		prd.done = false;
		prd.seed = seed;
		prd.depth = 0;
		prd.mouse_ray = 0;

		// Each iteration is a segment of the ray path.  The closest hit will
		// return new segments to be traced here.
		for (;;)
		{

			Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, sys_SceneEpsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);

			if (prd.done)
			{
				// We have hit the background or a luminaire
				prd.result += prd.radiance * prd.attenuation;
				break;
			}

			// Russian roulette termination 
			if (prd.depth >= rr_begin_depth)
			{
				float pcont = fmaxf(prd.attenuation);
				if (rnd(prd.seed) >= pcont || prd.depth > max_depth)
					break;
				prd.attenuation /= pcont;
			}

			prd.depth++;
			prd.result += prd.radiance * prd.attenuation;

			//if(prd.depth > max_depth)
			//   break;

			// Update ray data for the next path segment
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		}

		result += prd.result;
		seed = prd.seed;
	} while (--samples_per_pixel);

	//
	// Update the output buffer
	//
	float3 pixel_color = result / (sqrt_num_samples*sqrt_num_samples);
	if (frame_number > 1)
	{
		float a = 1.0f / (float)frame_number;
		float3 old_color = make_float3(sys_OutputBuffer[launch_index]);
		sys_OutputBuffer[launch_index] = make_float4(lerp(old_color, pixel_color, a), 1.0f);
	}
	else
	{
		sys_OutputBuffer[launch_index] = make_float4(pixel_color, 1.0f);
	}
}



//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------
#ifdef EMISSIVE_HIT

rtDeclareVariable(float3, emission_color, , );
rtDeclareVariable(float, t_hit_occluder, rtIntersectionDistance, );


RT_PROGRAM void diffuse_Emitter()
{
	current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
	current_prd.done = true;
}
#endif




//-----------------------------------------------------------------------------
//
//  Lambertian surface diffuse
//
//-----------------------------------------------------------------------------

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(int , cast_shadow_ray, , );

static __device__ void lambertian_surface_diffuse(float3 Kd, float3 ffnormal)
{
	Kd.x = clamp(Kd.x, 0.0f, 0.9f);
	Kd.y = clamp(Kd.y, 0.0f, 0.9f);
	Kd.z = clamp(Kd.z, 0.0f, 0.9f);
	
	float scene_epsilon = sys_SceneEpsilon;
	float3 hitpoint = ray.origin + t_hit * ray.direction;

	
	//
	// Generate a reflection ray.  This will be traced back in ray-gen.
	//
	current_prd.origin = hitpoint;

	//float z1 = rnd(current_prd.seed);
	//float z2 = rnd(current_prd.seed);
	//float3 p;
	//optix::cosine_sample_hemisphere(z1, z2, p);
	//optix::Onb onb(ffnormal);
	//onb.inverse_transform(p);
	//current_prd.direction = p;
	//

	float pdf;
	unitSquareToCosineHemisphere(rng2(current_prd.seed), ffnormal, current_prd.direction, pdf);


	
	// NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
	// with cosine density.
	if ( cast_shadow_ray == 0 && current_prd.depth > 0)
		current_prd.attenuation = current_prd.attenuation * Kd * (1.2f / current_prd.depth);
	else
	    current_prd.attenuation = current_prd.attenuation * Kd;
	current_prd.countEmitted = false;



	float3 result = make_float3(0.0f);
	if( cast_shadow_ray == 1)
	{
		unsigned int num_lights = lights.size();
		for (int i = 0; i < num_lights; ++i)
	    {
	    	ParallelogramLight light = lights[i];
		
		    const float z1 = 1.0f * rnd(current_prd.seed);
		    const float z2 = 1.0f * rnd(current_prd.seed);
		    const float3 light_pos = light.pos + light.v1 * z1 + light.v2 * z2;
    
    
		    // Calculate properties of light sample (for area based pdf)
		    const float  Ldist = optix::length(light_pos - hitpoint);
		    const float3 L = optix::normalize(light_pos - hitpoint);
		    const float  nDl = optix::dot(ffnormal, L);
		    const float  LnDl = optix::dot( light.normal , L);
    
    
		    // cast shadow ray
		    if (nDl > 0.0f && LnDl > 0.0f)
		    {
    
		    	PerRayData_shadow shadow_prd;
		    	shadow_prd.inShadow = false;
		    	// Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
		    	optix::Ray shadow_ray = optix::make_Ray(hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist);
		    	rtTrace(top_shadower, shadow_ray, shadow_prd);
		    	
		    	if (!shadow_prd.inShadow)
		    	{
		    		float size_factor = 1.0f;
		    		const float A = optix::length(optix::cross(light.v1 * size_factor , light.v2) * size_factor );
		    		// convert area based pdf to solid angle
		    		const float weight = nDl * LnDl * A / (M_PIf * Ldist * Ldist);
		    		result += light.color * weight;
		    	}
		    	
    
		    }
		

	    }
	}
	else
	{
	
		float3 env_light_pos = make_float3(100.0f,1000.0f, -100.0f);
		float3 env_light_normal = normalize(hitpoint - env_light_pos);
		const float3 L = normalize(env_light_pos - hitpoint);
		const float  nDl = dot(ffnormal, L);
		const float  LnDl = dot( env_light_normal , L);
		
		if(nDl > 0.0f)
		{
		    const float3 env_light_color = make_float3(1.0f);
			//const float  Ldist = optix::length(env_light_pos - hitpoint);
			const float  Ldist = 500.0f; // constant ?
		    
		    float size_factor = (20.0f);
			// const float A = optix::length(optix::cross(light.v1 * size_factor , light.v2) * size_factor ); // Area Light 
			const float A = 2.0f * M_PIf * size_factor * size_factor; 
		    const float weight = nDl  * A * fabsf(dot( ffnormal, normalize(sys_CameraW)))  / (M_PIf * Ldist);
			result += env_light_color * weight;
		}
		
	}

	
	
	current_prd.radiance = result;
	
}





rtDeclareVariable(int, geometry_id, , );
rtDeclareVariable(int, sys_focusedObject, , );
rtDeclareVariable(int, sys_selectedObject, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtBuffer<int> sys_SelectedObjectBuffer;

rtDeclareVariable(float3, Kd, , );
//-----------------------------------------------------------------------------
//
//  closest-hit
//
//-----------------------------------------------------------------------------
RT_PROGRAM void closest_hit()
{

	if (current_prd.mouse_ray == 1)
	{
		if (geometry_id >= 0)
		{
			sys_SelectedObjectBuffer[0] = geometry_id;
			current_prd.selected = geometry_id;
			return;

		}
		else
		{
			current_prd.selected = -1;
			return;
		}
	}
	else
	{
		if ((sys_focusedObject == geometry_id) || (sys_selectedObject == geometry_id) && geometry_id >= 0)
			current_prd.selected = geometry_id;
		else
			current_prd.selected = -1;
	}


	
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	lambertian_surface_diffuse(Kd, ffnormal);

}




rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, );
//-----------------------------------------------------------------------------
//
//  closest-hit Textured
//
//-----------------------------------------------------------------------------
RT_PROGRAM void closest_hit_textured()

{

	if (current_prd.mouse_ray == 1)
	{
		if (geometry_id >= 0)
		{
			sys_SelectedObjectBuffer[0] = geometry_id;
			current_prd.selected = geometry_id;
			return;

		}
		else
		{
			current_prd.selected = -1;
			return;
		}
	}
	else
	{
		if ((sys_focusedObject == geometry_id) || (sys_selectedObject == geometry_id) && geometry_id >= 0)
			current_prd.selected = geometry_id;
		else
			current_prd.selected = -1;
	}


	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	const float3 Kd_val = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
	lambertian_surface_diffuse(Kd_val, ffnormal);
	
}





//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------
rtDeclareVariable(PerRayData_shadow, current_prd_shadow, rtPayload, );
RT_PROGRAM void any_hit()
{
	current_prd_shadow.inShadow = true;
	rtTerminateRay();
}



//-----------------------------------------------------------------------------
//
//  Miss
//
//-----------------------------------------------------------------------------


rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(float3, bg_color2, , );
RT_PROGRAM void miss()
{
	

	current_prd.selected = -1;
	current_prd.miss = 1;
	
	current_prd.done = true;
	if (current_prd.depth == 0)
	{
	    float u = (float)launch_index.x / (float)launch_dim.x;
	    float v = (float)launch_index.y / (float)launch_dim.y;
	    float3 final_color = bg_color * u + bg_color2 * v;
	
		current_prd.radiance = final_color;
	}
	else
	{
	    float b = current_prd.depth/5.0f;
		float a = 1.0f - b;
		current_prd.radiance = a * make_float3(0.0f) + make_float3(0.25f)*b;
	}

}




//-----------------------------------------------------------------------------
//
//  Exception 
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bad_color, , );
RT_PROGRAM void exception()
{
	sys_OutputBuffer[launch_index] = make_float4(bad_color , 1.0f);
}