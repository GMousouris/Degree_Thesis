#include "hip/hip_runtime.h"
#pragma once

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "RT_app_config.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "random_number_generators.h"
#include "sampling_functions.h"

#include "light_structs.h"
#include "light_sample.h"

using namespace optix;




//rtDeclareVariable(Matrix4x4, sys_view_mat, , );
//rtDeclareVariable(Matrix4x4, sys_proj_mat, , );
rtDeclareVariable(Matrix4x4, sys_vp_mat, , );

rtDeclareVariable(int, sys_write_depth, , );
rtDeclareVariable(int, sys_only_write_depth, , );

rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(int, geometry_id, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );

rtDeclareVariable(float,    sys_SceneEpsilon, , );

rtDeclareVariable(int, sys_NumLights, , );
rtBuffer<LightDefinition> sys_LightDefinitions;
rtDeclareVariable(int, cast_shadow_ray, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, ray,                  rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance,  rtIntersectionDistance, );

rtDeclareVariable(PerRayData_c, thePrd,                  rtPayload, );

 

rtDeclareVariable(float3, varGeoNormal, attribute geometric_normal, );
rtDeclareVariable(float3, varNormal, attribute shading_normal, );




//


rtDeclareVariable(int, sys_cast_reflections, , );
rtDeclareVariable(int, sys_use_fresnel_equations, , );
rtDeclareVariable(unsigned int, max_depth, , );
rtDeclareVariable(float3, sys_CameraW, ,);
rtDeclareVariable(float3, sys_CameraPosition, ,);
rtDeclareVariable(unsigned int, sys_sample_glossy_index, ,);
rtDeclareVariable(unsigned int, sys_sample_light_refl, ,);
rtDeclareVariable(unsigned int, sys_bsdf_light_refl, ,);
rtDeclareVariable(int, sys_const_light_model, , );

rtDeclareVariable(int, is_light, , );
rtDeclareVariable(int, is_sampler, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, ); // temp //

RT_FUNCTION void lambertian_surface_sample(float3 const& Kd, float3 const& Kr, float3 const& Ka, float const& exp )
{
  State state; // All in world space coordinates!

  state.geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
  state.normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

  thePrd.pos      = ray.origin + ray.direction * theIntersectionDistance; // Advance the path to the hit position in world coordinates.
  thePrd.distance = theIntersectionDistance; // Return the current path segment distance, needed for absorption calculations in the integrator.


  if ( sys_write_depth == 1 && thePrd.depth == 0 )
  {
        float4 p      = make_float4(thePrd.pos.x, thePrd.pos.y, thePrd.pos.z, 1.0f);
	    float4 proj_p = sys_vp_mat * p;
	    float3 ndc_p  = make_float3(proj_p.x,proj_p.y,proj_p.z) / proj_p.w;
	    float  depth  = ndc_p.z;

	    float f_depth;
	    f_depth = depth * 0.5f + 0.5f;
	    thePrd.depth_value = f_depth;
  }




  thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, state.geoNormal)) ? FLAG_FRONTFACE : 0;
  if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
  {
      state.geoNormal = -state.geoNormal;
      state.normal    = -state.normal;
  }

  
  thePrd.radiance   = make_float3(0.0f);
  thePrd.f_over_pdf = make_float3(0.0f);
  thePrd.pdf        = 0.0f;
  thePrd.flags      = (thePrd.flags & ~FLAG_DIFFUSE); // 

  float refl       = Kr.x;
  float ior        = Ka.x;
  float rand_value = 1.0f;
  float pattern    = 1.0f;

  if ( sys_cast_reflections == 1 &&  geometry_id != -100)
  {
     rand_value = rng2( thePrd.seed ).x;
  }

  float fresn = sys_use_fresnel_equations == 1? fresnel(thePrd.wi, state.normal, ior, refl) : 1.0f;
  
  
  int DIFFUSE_IND = 0;
  if (  refl  >= rand_value 
     && sys_cast_reflections == 1 )
  {
     
     if      ( sys_sample_glossy_index == 0 )
        sample_bsdf_specular_reflection(Kd, refl, exp, state, thePrd);
     else if ( sys_sample_glossy_index == 1 )
	    sample_bsdf_specular_glossy_reflection  ( Kd, refl, ior, exp,  state, thePrd );
     else if ( sys_sample_glossy_index == 2 )
	    sample_bsdf_specular_glossy_reflection2 ( Kd, refl, ior, exp,  state, thePrd );
     else if ( sys_sample_glossy_index == 3 )
        sample_bsdf_specular_glossy_reflection2 ( Kd, refl, ior, exp,  state, thePrd );

	 if( sys_use_fresnel_equations == 1 )
	      thePrd.f_over_pdf *= fresn;

  }
  else
  {
     DIFFUSE_IND = 1;
	   sample_bsdf_diffuse_reflection( Kd, state, thePrd );
     if(    sys_cast_reflections == 1 
         && geometry_id != -100
	       && sys_use_fresnel_equations == 1 )
		 thePrd.f_over_pdf *= (1.0f - fresn);

  }

  

#if USE_NEXT_EVENT_ESTIMATION
  
  if ( ( DIFFUSE_IND == 1 ) && ( 0 < sys_NumLights && cast_shadow_ray == 1 ) )
  {
      // generate a random float2 //
      const float2 sample    = rng2(thePrd.seed); 
      const float3 sample3f  = make_float3( sample.x, sample.y, rng(thePrd.seed) );

      int num_lights = 0;
      for(int i = 0; i< sys_NumLights; i++)
      {
          if(sys_LightDefinitions[i].is_on) 
          {
		      num_lights++;
          }  
      }
      
	  
	  
      LightSample lightSample;
	  LightDefinition rand_light;
      lightSample.index          = optix::clamp( static_cast<int>(floorf(rng(thePrd.seed) * sys_NumLights)), 0, sys_NumLights - 1 );  // picking a random light
	  rand_light = sys_LightDefinitions[ lightSample.index ];
	  sample_Light( 
	       thePrd.pos,
           sample3f,
		   lightSample,
		   rand_light,
		   num_lights);
	  
      if ( lightSample.pdf > 0.0f ) // Useful light sample?
      {
      
         float4 bsdf_pdf;
         bsdf_pdf = ( DIFFUSE_IND == 1 ) ? eval_bsdf_diffuse_reflection(  Kd, state, thePrd,       lightSample.direction ) 
                                         : eval_bsdf_specular_reflection( Kd, refl, state, thePrd, lightSample.direction );
         
         if ( 0.0f < bsdf_pdf.w && isNotNull(make_float3(bsdf_pdf)) )
         {
        
             PerRayData_shadow_c prdShadow;
             prdShadow.visible = true; 
             optix::Ray ray    = optix::make_Ray(thePrd.pos, lightSample.direction, 1, sys_SceneEpsilon, lightSample.distance - sys_SceneEpsilon); // Shadow ray.
             rtTrace( top_shadower , ray, prdShadow );

             if ( prdShadow.visible )
             {
                 const float weight = powerHeuristic( lightSample.pdf, bsdf_pdf.w );
                 thePrd.radiance   += pattern 
                  * make_float3(bsdf_pdf)
                  * lightSample.emission 
                  * ( weight * optix::dot(lightSample.direction, state.normal) / lightSample.pdf );
             }
        }
    }

  }
  else if ( cast_shadow_ray == 0 && DIFFUSE_IND == 1 && max_depth > 1 && geometry_id != -100 )
  {

    const float2 sample = rng2(thePrd.seed);
    LightSample lightSample; // Sample one of many lights. 
    //sample_light_constant(thePrd.pos, sample, lightSample, 1);
	  sample_light_camera_constant(thePrd.pos, sys_CameraW, sample, lightSample, 1);

    float4 bsdf_pdf;
	  bsdf_pdf = eval_bsdf_diffuse_reflection(Kd, state, thePrd, lightSample.direction );
    const float weight = powerHeuristic(lightSample.pdf, bsdf_pdf.w);
	  float dot_weight = dot(lightSample.direction, state.normal);
	  dot_weight = clamp(dot_weight, 0.1f, 100.0f);
	  thePrd.radiance += pattern * make_float3(bsdf_pdf) * lightSample.emission * ( weight * dot_weight / lightSample.pdf );
    

  }
  else if ( cast_shadow_ray == 0 && DIFFUSE_IND == 1 && max_depth == 1 || geometry_id == -100 )
  {   

 
	  if (sys_const_light_model == 0)
	  {
	      
		  float const_dist = 150.0f;
		  float3 light_pos = thePrd.pos - sys_CameraW * const_dist;
		  float3 light_dir = normalize(sys_CameraPosition - thePrd.pos);
		  float3 viewDir = normalize(sys_CameraPosition - thePrd.pos);
		  float3 halfV = normalize(viewDir + light_dir);
		  float NdotL   = fabsf(dot(light_dir, state.normal));
		  
		  float3 fade  = make_float3(1.0f);
		  float fade_s = 0.45f;
		  if(    geometry_id != -100
		      && NdotL < fade_s 
			  )
		  {
		      float t = (fade_s - NdotL) / fade_s;
			  t       = clamp(t, 0.0f, 1.0f);

			  fade.x = t*(0.0f) + (1.0f - t)*1.0f;
			  fade.y = t*(0.0f) + (1.0f - t)*1.0f;
			  fade.z = t*(0.0f) + (1.0f - t)*1.0f;
		  }
		  float NdotH = fabsf(dot(halfV, state.normal));

		  float spotEffect;
		  spotEffect  = 1.f * (dot(sys_CameraW, -state.normal));
		  spotEffect *= spotEffect;
		  spotEffect  = clamp(spotEffect, 0.07f, 100.0f); // 0.1 - 100.0f

		  float3 irr       = spotEffect * NdotL / const_dist * make_float3(1.0f);
		  float3 diff_refl = irr * Kd;
		  float specularNorm = (0.01f + 8.0f) / (8.0f * M_PIf);

		  float3 spec_refl = make_float3(0.0f);
		  if (NdotL > 0.0f)
			  spec_refl = irr * specularNorm * powf(NdotH, 10.0f + 0.001f);
		  thePrd.radiance += pattern * 2500.0f * diff_refl * fade + spec_refl;
	  }
	  else if (sys_const_light_model == 1)
	  {
		  float3 specular;
		  float phong_exp;
		  float Ldist;
		  float3 emission;

		  if( geometry_id == -100 )
		  {
			 phong_exp = 10.0f;
			 Ldist     = 5000.0f;
			 emission  = make_float3(0.7f);
			 specular  = make_float3(0.2f);
		  }
		  else
		  {
		     phong_exp = 10.0f;
			 Ldist     = 4000.0f;
			 emission  = make_float3(0.9f);
			 specular  = make_float3(0.5f);
		  }
		  
		  float3 L = normalize(sys_CameraPosition - thePrd.pos);
		  float nDl = dot(state.normal, L);
		  //nDl *= nDl;
		  //nDl  = clamp(nDl, 0.0f, 1.0f);
		  float d = dot(sys_CameraW, -state.normal);
		  //d = clamp(d, 0.5f, 1.0f);
		  d = 1.0f;

		  float3 Lc        = emission;
		  thePrd.radiance += pattern * Kd * nDl * d * Lc;
		  float3 H = normalize(L - ray.direction);
		  float nDh = dot(state.normal, H);

		  if (nDh > 0)
		  {
			  float power = pow(nDh, phong_exp);
			  thePrd.radiance += specular * power * Lc;
		  }

	  }

	 
  }
    
  
#endif // USE_NEXT_EVENT_ESTIMATION

}

















RT_FUNCTION void closest_hit_Light( float3 const& Kd )
{ 
     
     LightDefinition light;
     for(int i = 0; i< sys_NumLights; i++)
	 {
	      if( sys_LightDefinitions[i].id == geometry_id )
		  {
		       light = sys_LightDefinitions[i];
			   break;
		  }
	 }

     thePrd.pos      = ray.origin + ray.direction * theIntersectionDistance;
     thePrd.distance = theIntersectionDistance; 

     const float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal)); // PERF Not really needed when it's know that light geometry is not under Transforms.
     thePrd.radiance = make_float3(0.0f);

	   const float cosTheta = optix::dot(thePrd.wo, geoNormal);
	   thePrd.flags |= (0.0f <= cosTheta) ? FLAG_FRONTFACE : 0;

     if (thePrd.flags & FLAG_FRONTFACE)
     {
         
		 thePrd.radiance = make_float3(1.0f);
		 //thePrd.radiance = light.emission;

		 float light_area = light.area;
         const float pdfLight = (thePrd.distance * thePrd.distance) / ( light_area * cosTheta); 
         if ((thePrd.flags & FLAG_DIFFUSE) && DENOMINATOR_EPSILON < pdfLight)
         {
              thePrd.radiance *= powerHeuristic(thePrd.pdf, pdfLight);
         }

      }

      thePrd.flags |= FLAG_TERMINATE;
}

RT_FUNCTION void closest_hit_Sampler(float3 const& kd)
{

}










rtDeclareVariable(int, sys_focusedObject, , );
rtDeclareVariable(int, sys_selectedObject, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtBuffer<int> sys_SelectedObjectBuffer;

rtBuffer<float3>       sys_mouseHit_buffer_Output;
rtBuffer<float>        sys_custom_dir_Output;

rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Kr, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float, phong_exp, , );

rtDeclareVariable(unsigned int, sys_abort, , );
rtDeclareVariable(int,    sys_is_custom_dir , ,);



rtDeclareVariable(float3, sys_m_cameraW, , );


//-----------------------------------------------------------------------------
//
//  closest-hit
//
//-----------------------------------------------------------------------------
RT_PROGRAM void closest_hit()
{
   
  if (thePrd.mouse_ray == 1)
  {
    if (geometry_id >= 0)
    {
      sys_SelectedObjectBuffer[0] = geometry_id;
      thePrd.selected = geometry_id;

	  
      float3 normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
	  
	  float3 o   =  ray.origin;//rtTransformNormal(RT_OBJECT_TO_WORLD, ray.origin);
	  float3 dir =  ray.direction; //rtTransformNormal(RT_OBJECT_TO_WORLD, ray.direction);
	  
	  float3 pos       = o + dir * theIntersectionDistance; 
	  sys_mouseHit_buffer_Output[0] = pos;
	  sys_mouseHit_buffer_Output[1] = normal;
	  sys_mouseHit_buffer_Output[2] = make_float3(1.0f);
	  
	  //
	  
      return;

    }
    else
    {
      thePrd.selected = -1;
      return;
    }
  }
  else
  {
    if ((sys_focusedObject == geometry_id) || (sys_selectedObject == geometry_id) && geometry_id >= 0)
      thePrd.selected = geometry_id;
    else
      thePrd.selected = -1;
  }
 
  if( is_light == 1 )
     closest_hit_Light( Kd );
  else
     lambertian_surface_sample(Kd, Kr, Ka, phong_exp);
}




rtTextureSampler<float4, 2> Kd_map;
//-----------------------------------------------------------------------------
//
//  closest-hit Textured
//
//-----------------------------------------------------------------------------
RT_PROGRAM void closest_hit_textured()
{
    
  if (thePrd.mouse_ray == 1)
  { 
    if (geometry_id >= 0)
    {
        
      sys_SelectedObjectBuffer[0] = geometry_id;
      thePrd.selected = geometry_id;

      float3 normal      = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
	  
	  float3 pos       = ray.origin + ray.direction * theIntersectionDistance; 
	  sys_mouseHit_buffer_Output[0] = pos;
	  sys_mouseHit_buffer_Output[1] = normal;
	  sys_mouseHit_buffer_Output[2] = make_float3(1.0f);

      return;

    }
    else
    {
      thePrd.selected = -1;
      return;
    }
  }
  else
  {
    if ((sys_focusedObject == geometry_id) || (sys_selectedObject == geometry_id) && geometry_id >= 0)
      thePrd.selected = geometry_id;
    else
      thePrd.selected = -1;
  }


  const float3 Kd_val = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));
  lambertian_surface_sample(Kd_val, Kr, Ka,  phong_exp);
  
}



