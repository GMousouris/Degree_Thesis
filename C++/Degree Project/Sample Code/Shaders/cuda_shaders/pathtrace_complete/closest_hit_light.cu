
#include <hip/hip_runtime.h>
/*

#include "RT_app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>


#include "per_ray_data.h"
#include "light_structs.h"
#include "shader_common.h"

rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(optix::Ray, theRay,                  rtCurrentRay, );
rtDeclareVariable(float,      theIntersectionDistance, rtIntersectionDistance, );
rtDeclareVariable(PerRayData, thePrd,                  rtPayload, );
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );


rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    parLightIndex, , );  // Index into the sysLightDefinitions array.

// Very simple closest hit program just for rectangle area lights.
RT_PROGRAM void closesthit_light()
{

  thePrd.pos      = theRay.origin + theRay.direction * theIntersectionDistance; // Advance the path to the hit position in world coordinates.
  thePrd.distance = theIntersectionDistance; // Return the current path segment distance, needed for absorption calculations in the integrator.

  const float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal)); // PERF Not really needed when it's know that light geometry is not under Transforms.

  const float cosTheta = optix::dot(thePrd.wo, geoNormal);
  thePrd.flags |= (0.0f <= cosTheta) ? FLAG_FRONTFACE : 0;

  thePrd.radiance = make_float3(0.0f); // Backside is black.

  if (thePrd.flags & FLAG_FRONTFACE) // Looking at the front face?
  {
    const LightDefinition light = sysLightDefinitions[parLightIndex];
    thePrd.radiance = light.emission;

    const float pdfLight = (thePrd.distance * thePrd.distance) / (light.area * cosTheta); // Solid angle pdf. Assumes light.area != 0.0f.
    if ((thePrd.flags & FLAG_DIFFUSE) && DENOMINATOR_EPSILON < pdfLight)
    {
      thePrd.radiance *= powerHeuristic(thePrd.pdf, pdfLight);
    }

  }

  thePrd.flags |= FLAG_TERMINATE;
}


*/