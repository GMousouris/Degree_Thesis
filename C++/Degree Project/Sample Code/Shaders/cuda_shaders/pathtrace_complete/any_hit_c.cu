#include <optix.h>
#include "per_ray_data.h"

using namespace optix;



rtDeclareVariable(PerRayData_shadow_c, thePrdShadow, rtPayload, );
RT_PROGRAM void any_hit()
{
    thePrdShadow.visible = false;
    rtTerminateRay();
}
