#include "hip/hip_runtime.h"
#pragma once

#include <optixu/optixu_math_namespace.h>

#include "random_number_generators.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "RT_app_config.h"

using namespace optix;

rtDeclareVariable(int, sys_write_depth, , );
rtDeclareVariable(int, sys_only_write_depth, , );

rtDeclareVariable(float3, sys_CameraPosition, , );
rtDeclareVariable(float3, sys_CameraU, , );
rtDeclareVariable(float3, sys_CameraV, , );
rtDeclareVariable(float3, sys_CameraW, , );
rtDeclareVariable(float, sys_SceneEpsilon, , );

rtBuffer<float, 2>                 sys_DepthBuffer;
rtBuffer<float4, 2>                sys_OutputBuffer;
//rtBuffer<float, 2>               sys_MissOutputBuffer;              
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

rtDeclareVariable(unsigned int, frame_number, , );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );

rtDeclareVariable(unsigned int, min_depth, , );
rtDeclareVariable(unsigned int, max_depth, , );

rtDeclareVariable(unsigned int , sys_abort, , );


//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------


RT_FUNCTION void integrator(PerRayData_c& prd, float3& radiance)
{

  radiance          = make_float3(0.0f); // Start with black.
  float3 throughput = make_float3(1.0f); // The throughput for the next radiance, starts with 1.0f.
  prd.flags         = 0;
  prd.flags        &= FLAG_CLEAR_MASK;   // Clear all non-persistent flags. In this demo only the last diffuse surface interaction stays.
  int depth         = 0;

  while ( prd.depth < max_depth)
  {

    prd.wo        = -prd.wi;           // Direction to observer.
    optix::Ray ray = optix::make_Ray(prd.pos, prd.wi, 0, sys_SceneEpsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);

    radiance += throughput * prd.radiance;

    // Path termination by miss shader or sample() routines.
    // If terminate is true, f_over_pdf and pdf might be undefined.
    if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
    {
      break;
    }

    // PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
    throughput *= prd.f_over_pdf;

    // Unbiased Russian Roulette path termination.
    if ( min_depth <= prd.depth) // Start termination after a minimum number of bounces.
    {
      const float probability = fmaxf(throughput); // DAR Other options: // intensity(throughput); // fminf(0.5f, intensity(throughput));
      if (probability < rng(prd.seed)) // Paths with lower probability to continue are terminated earlier.
      {
        break;
      }
      throughput /= probability; // Path isn't terminated. Adjust the throughput so that the average is right again.
    }

    ++prd.depth; // Next path segment.
	//++depth;
  }
}







RT_PROGRAM void path_trace()
{

  PerRayData_c prd;
  prd.depth = 0;
  prd.mouse_ray = 0;
  prd.miss = 0;
  
  // Initialize the random number generator seed from the linear pixel index and the iteration index.
  prd.seed = tea2<8>(launch_index.y * launch_dim.x + launch_index.x, frame_number);
  
  //float2 jitter = make_float2(0.0f);
  //if (max_depth > 1)

  float2 jitter   = rng2(prd.seed);
  //float2 jitter = make_float2(0.0f);

  const float2 pixel = make_float2(launch_index);
  const float2 fragment = pixel + jitter; // Random jitter of the fragment location in this pixel.
  const float2 screen = make_float2(launch_dim);
  const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

  // The integrator expects the next path segments ray.origin in prd.pos and the next ray.direction in prd.wi.
  prd.pos = sys_CameraPosition;
  prd.wi  = optix::normalize(ndc.x * sys_CameraU + ndc.y * sys_CameraV + sys_CameraW);

  float3 radiance;
  integrator(prd, radiance); // In this case a unidirectional path tracer.

#if USE_DEBUG_EXCEPTIONS
  // DAR DEBUG Highlight numerical errors.
  if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
  {
    radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
  }
  else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
  {
    radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
  }
  else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
  {
    radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
  }
#else
  // NaN values will never go away. Filter them out before they can arrive in the output buffer.
  // This only has an effect if the debug coloring above is off!
  if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
  {
    

	if ( sys_write_depth == 1 )
	{
	    sys_DepthBuffer[launch_index] = prd.depth_value;
	}

	if ( sys_only_write_depth == 1 )
	{
	    return;
  }
   
  float4 dst    = sys_OutputBuffer[launch_index]; 
  float4 output = ( frame_number > 1 )? optix::lerp(dst, make_float4(radiance, 1.0f), 1.0f / (float) (frame_number + 1)) 
                                        : make_float4(radiance, 1.0f);
  
  //output.x = output.x > 1 ? 1: output.x;
  //output.y = output.y > 1 ? 1: output.y;
  //output.z = output.z > 1 ? 1: output.z;
  sys_OutputBuffer[launch_index] = output;
  
	
  }
}




