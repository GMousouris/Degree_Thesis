#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "RT_app_config.h"
#include "per_ray_data.h"
#include "shader_common.h"


rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(float3, bg_color2, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

rtDeclareVariable(PerRayData_c, thePrd, rtPayload, );
rtDeclareVariable(int, cast_shadow_ray, , );

rtBuffer<float3>       sys_mouseHit_buffer_Output;


RT_PROGRAM void miss()
{
    if(thePrd.mouse_ray == 1)
	{
	    sys_mouseHit_buffer_Output[2] = make_float3(0.0f);
	}
	
	thePrd.selected = -1;
	
	if (thePrd.depth == 0)
	{
	    thePrd.miss = 1;
	    thePrd.depth_value = 1;
		//float u = (float)launch_index.x / (float)launch_dim.x;
	    float v = (float)launch_index.y / (float)launch_dim.y;

		
		float3 final_color = bg_color * v + bg_color2 * (1.0f - v);
		thePrd.radiance    = final_color;
	}
	else
	{
	    if( cast_shadow_ray == 1 )
		    thePrd.radiance = make_float3(0.0f);
		else
		   thePrd.radiance = make_float3(1.0f);
		    
	}

    thePrd.flags |= FLAG_TERMINATE;



}
