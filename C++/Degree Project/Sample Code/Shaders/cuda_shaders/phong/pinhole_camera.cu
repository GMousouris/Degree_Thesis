#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <common.h>
#include "helpers.h"
#include "per_ray_data.h"

using namespace optix;


rtDeclareVariable(float3,        sys_CameraPosition, , );
rtDeclareVariable(float3,        sys_CameraU, , );
rtDeclareVariable(float3,        sys_CameraV, , );
rtDeclareVariable(float3,        sys_CameraW, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         sys_SceneEpsilon, , );

rtDeclareVariable(float2,        sys_mousePos, , );


rtBuffer<float4, 2>              sys_OutputBuffer;
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;
rtDeclareVariable(int, geometry_id, , );
rtDeclareVariable(int, sys_selectedObject, , );
rtDeclareVariable(int, sys_highlightPass, , );
rtDeclareVariable(int, sysIterationIndex, , );


//#define TIME_VIEW

RT_PROGRAM void pinhole_camera()
{



#ifdef TIME_VIEW
  clock_t t0 = clock(); 
#endif

  float scene_epsilon = sys_SceneEpsilon;
  float3 ray_origin;
  float3 ray_direction;

#define METHOD1
#ifdef METHOD1

  const float2 pixel = make_float2(launch_index);
  const float2 fragment = pixel + make_float2(0.5f);
  const float2 screen = make_float2(launch_dim);
  const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

  ray_origin = sys_CameraPosition;
  ray_direction = optix::normalize(ndc.x * sys_CameraU + ndc.y * sys_CameraV + sys_CameraW);

#endif

//#define METHOD2
#ifdef METHOD2

  float3 eye = sys_CameraPosition;
  float3 U   = sys_CameraU;
  float3 V   = sys_CameraV;
  float3 W   = sys_CameraW;
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;

  ray_origin = sys_CameraPosition;
  ray_direction = normalize(d.x*sys_CameraU + d.y*sys_CameraV + sys_CameraW);

#endif



 
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;
  prd.mouse_ray = 0;
  prd.selected = 0;
  prd.miss = 0;


  rtTrace(top_object, ray, prd);


#ifdef TIME_VIEW
  clock_t t1 = clock(); 
 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  output_buffer[launch_index] = make_color( make_float3(  pixel_time ) ); 
#else

  float a;

  if (prd.selected == 1)
	  a = 1.0f;
  else if (prd.selected == 0)
	  a = 0.0f;
  else
	  a = 0.0f;
  
  if (sys_highlightPass == 0)
	  sys_OutputBuffer[launch_index] = make_float4(prd.radiance, prd.selected);
 
#endif
}