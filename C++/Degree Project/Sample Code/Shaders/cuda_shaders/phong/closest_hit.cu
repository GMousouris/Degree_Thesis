#include "hip/hip_runtime.h"
#pragma once
#include "closest_hit_functions.h"


using namespace optix;

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kr, , );
rtDeclareVariable(float, phong_exp, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, geometry_id, , );
rtDeclareVariable(int, sys_focusedObject, , );
rtDeclareVariable(int, sys_selectedObject, , );



RT_PROGRAM void closest_hit()
{

	
	if (prd.mouse_ray == 1)
	{
		if (geometry_id >= 0)
		{ 
			sys_SelectedObjectBuffer[0] = geometry_id;
			prd.selected = geometry_id;
			return;
			
		}
		else
		{
			prd.selected = -1;
			return;
		}
	}
	else
	{
		if ((sys_focusedObject == geometry_id) || (sys_selectedObject == geometry_id) && geometry_id >= 0)
			prd.selected = geometry_id;
		else
			prd.selected = -1;
	}

	

	
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

	
	phong_closest_hit(Kd, Ka, Ks, Kr, phong_exp, ffnormal);
}


rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable(float3, texcoord, attribute texcoord, );

RT_PROGRAM void closest_hit_textured()
{

	if (prd.mouse_ray == 1)
	{
		if (geometry_id >= 0)
		{
			sys_SelectedObjectBuffer[0] = geometry_id;
			prd.selected = geometry_id;
			return;
		}
		else
		{
			prd.selected = -1;
			return;
		}
	}
	else
	{
		if ((sys_focusedObject == geometry_id) || (sys_selectedObject == geometry_id) && geometry_id >= 0)
		{
			prd.selected = geometry_id;
		}
		else
		{
			prd.selected = -1;
		}
	}


	
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
	const float3 Kd_val = make_float3(tex2D(Kd_map, texcoord.x, texcoord.y));

	
	phong_closest_hit(Kd_val, Ka, Ks, Kr, phong_exp, ffnormal);
	
}
