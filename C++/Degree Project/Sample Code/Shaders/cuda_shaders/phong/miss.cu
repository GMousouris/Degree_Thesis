#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "per_ray_data.h"

using namespace optix;




rtDeclareVariable(PerRayData_radiance, current_prd, rtPayload, ,);
rtDeclareVariable(float3, bg_color, , );
rtDeclareVariable(float3, bg_color2, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );



RT_PROGRAM void miss()
{
	
	
	/* experimenting bg color */
	float u = (float)launch_index.x / (float)launch_dim.x;
	float v = (float)launch_index.y / (float)launch_dim.y;
	float3 final_color = bg_color * u + bg_color2 * v;
	
	//current_prd.result = final_color;
	current_prd.selected = -1;
	current_prd.miss = 1;
	
	current_prd.done = true;
	if (current_prd.depth == 0)
	{
		current_prd.radiance = final_color;
	}
	else
	{
	    float b = current_prd.depth/5.0f;
		float a = 1.0f - b;
		current_prd.radiance = a * make_float3(0.0f) + make_float3(0.25f)*b;
	}

	

}
