#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <common.h>
#include "helpers.h"
#include "per_ray_data.h"

using namespace optix;




rtDeclareVariable(float3, bad_color, , );
rtBuffer<float4, 2>                sys_OutputBuffer;
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


RT_PROGRAM void exception()
{
	sys_OutputBuffer[launch_index] = make_float4(bad_color , 1.0f);
}