#include "hip/hip_runtime.h"


#include "RT_app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "per_ray_data.h"
#include "light_definition.h"
#include "shader_common.h"
#include "sampling_functions.h"

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    sysNumLights, , ); // PERF Used many times and faster to read than sysLightDefinitions.size().




// Note that all light sampling routines return lightSample.direction and lightSample.distance in world space!

RT_CALLABLE_PROGRAM void sample_light_constant(float3 const& point, const float2 sample, LightSample& lightSample)
{
  unitSquareToSphere(sample.x, sample.y, lightSample.direction, lightSample.pdf);

  // Environment lights do not set the light sample position!
  lightSample.distance = RT_DEFAULT_MAX; // Environment light.

  // Explicit light sample. White scaled by inverse probabilty to hit this light.
  lightSample.emission = make_float3(sysNumLights);
}


RT_CALLABLE_PROGRAM void sample_light_parallelogram(float3 const& point, const float2 sample, LightSample& lightSample)
{
  lightSample.pdf = 0.0f; // Default return, invalid light sample (backface, edge on, or too near to the surface)

  const LightDefinition light = sysLightDefinitions[lightSample.index]; // The light index is picked by the caller!

  lightSample.position  = light.position + light.vecU * sample.x + light.vecV * sample.y; // The light sample position in world coordinates.
  lightSample.direction = lightSample.position - point; // Sample direction from surface point to light sample position.
  lightSample.distance  = optix::length(lightSample.direction);
  if (DENOMINATOR_EPSILON < lightSample.distance)
  {
    lightSample.direction /= lightSample.distance; // Normalized direction to light.
 
    const float cosTheta = optix::dot(-lightSample.direction, light.normal);
    if (DENOMINATOR_EPSILON < cosTheta) // Only emit light on the front side.
    {
      // Explicit light sample, must scale the emission by inverse probabilty to hit this light.
      lightSample.emission = light.emission * float(sysNumLights); 
      lightSample.pdf      = (lightSample.distance * lightSample.distance) / (light.area * cosTheta); // Solid angle pdf. Assumes light.area != 0.0f.
    }
  }
}
