#include "hip/hip_runtime.h"

#include <optix_world.h>
#include <common.h>
#include "helpers.h"
#include "per_ray_data.h"

using namespace optix;



rtDeclareVariable(float3, sys_m_CameraPosition, , );
rtDeclareVariable(float3, sys_m_CameraU, , );
rtDeclareVariable(float3, sys_m_CameraV, , );
rtDeclareVariable(float3, sys_m_CameraW, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(float,  sys_SceneEpsilon, , );

rtDeclareVariable(float2, sys_mousePos, , );
rtDeclareVariable(float3, sys_cast_dir, , );
rtDeclareVariable(int,    sys_is_custom_dir , ,);

rtDeclareVariable(int,    sys_utility_ray, ,);
rtDeclareVariable(float3, sys_utility_ray_origin, ,);
rtDeclareVariable(float3, sys_utility_ray_direction, ,);


rtBuffer<float4, 2>                sys_OutputBuffer;
//rtBuffer<uchar4, 2>              sys_OutputBuffer;
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );
rtDeclareVariable(uint,  sys_m_width, , );
rtDeclareVariable(uint,  sys_m_height, , );

rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;
rtDeclareVariable(int, geometry_id, , );

rtDeclareVariable(unsigned int, shader_index, ,);

RT_PROGRAM void mouse_ray_generation()
{
    
      optix::uint2 dim = optix::make_uint2(sys_m_width, sys_m_height);
	  const float2 pixel = sys_mousePos;
	  const float2 fragment = pixel + make_float2(0.5f);
	  const float2 screen = make_float2(dim);
	  const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

	  float3 ray = make_float3(ndc.x, ndc.y, 1.0f);
	  float3 U = sys_m_CameraU;
	  float3 V = sys_m_CameraV;
	  float3 W = sys_m_CameraW;

	   
	  float3 ray_origin;
	  float3 ray_dir;

	  if( sys_utility_ray == 0 )
	  {
	     ray_origin = sys_m_CameraPosition;
	     ray_dir = normalize(ndc.x * U - ndc.y * V + W);
	  }
	  else
	  {
	     ray_origin = sys_utility_ray_origin;
		 ray_dir    = sys_utility_ray_direction;
		 
	  }

	  optix::Ray mouse_ray = optix::make_Ray(ray_origin, ray_dir, RADIANCE_RAY_TYPE, sys_SceneEpsilon, RT_DEFAULT_MAX);
	  if( shader_index == 0 || shader_index == 1 )
      {
	     PerRayData_radiance prd;
	     prd.depth = 0;
	     prd.mouse_ray = 1;
	     prd.selected = 0;
	     rtTrace(top_object, mouse_ray, prd);
	  }
	  else if ( shader_index == 2 )
      {
	     PerRayData_c prd;
	     prd.depth     = 0;
	     prd.mouse_ray = 1;
	     prd.selected  = 0;
	     rtTrace(top_object, mouse_ray, prd);
	  }
	


}

